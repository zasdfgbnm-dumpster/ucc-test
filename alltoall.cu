#include <condition_variable>
#include <deque>
#include <fstream>
#include <iostream>
#include <memory>
#include <mutex>
#include <queue>
#include <string>
#include <thread>
#include <chrono>
#include <filesystem>

#include <ucc/api/ucc.h>
#include <ucp/api/ucp.h>

#include "utils.hpp"

using T = int;
ucc_datatype_t dtype = UCC_DT_FLOAT32;

extern const int N;
extern int world_size;
extern int rank;
const int size_ = 5; // TODO: what is this?

void check_cuda(hipError_t);
int get_device();
void set_device(int i);

constexpr auto kUnsetTimeout = std::chrono::milliseconds(-1);

enum OpType { ALLTOALL_BASE };

bool isP2POp(OpType) { return true; }

#define TORCH_UCC_DEVICE_NOT_SET -2

#define TORCH_UCX_COMM_BITS 15
#define TORCH_UCX_RANK_BITS 16
#define TORCH_UCX_TAG_BITS 32
#define TORCH_UCX_OOB_BITS 1

#define TORCH_UCX_COMM_BITS_OFFSET 0
#define TORCH_UCX_RANK_BITS_OFFSET TORCH_UCX_COMM_BITS
#define TORCH_UCX_TAG_BITS_OFFSET (TORCH_UCX_COMM_BITS + TORCH_UCX_RANK_BITS)
#define TORCH_UCX_OOB_BITS_OFFSET                                              \
  (TORCH_UCX_COMM_BITS + TORCH_UCX_RANK_BITS + TORCH_UCX_TAG_BITS)

#define TORCH_UCX_MAX_COMM ((((uint64_t)1) << TORCH_UCX_COMM_BITS) - 1)
#define TORCH_UCX_MAX_RANK ((((uint64_t)1) << TORCH_UCX_RANK_BITS) - 1)
#define TORCH_UCX_MAX_TAG ((((uint64_t)1) << TORCH_UCX_TAG_BITS) - 1)
#define TORCH_UCX_MAX_OOB ((((uint64_t)1) << TORCH_UCX_OOB_BITS) - 1)

#define TORCH_UCX_COMM_MASK (TORCH_UCX_MAX_COMM << TORCH_UCX_COMM_BITS_OFFSET)
#define TORCH_UCX_RANK_MASK (TORCH_UCX_MAX_RANK << TORCH_UCX_RANK_BITS_OFFSET)
#define TORCH_UCX_TAG_MASK (TORCH_UCX_MAX_TAG << TORCH_UCX_TAG_BITS_OFFSET)
#define TORCH_UCX_OOB_MASK (TORCH_UCX_MAX_OOB << TORCH_UCX_OOB_BITS_OFFSET)

#define TORCH_UCX_MAKE_P2P_TAG(_tag, _rank, _comm)                             \
  ((((uint64_t)(_tag)) << TORCH_UCX_TAG_BITS_OFFSET) |                         \
   (((uint64_t)(_rank)) << TORCH_UCX_RANK_BITS_OFFSET) |                       \
   (((uint64_t)(_comm)) << TORCH_UCX_COMM_BITS_OFFSET))

#define TORCH_UCX_MAKE_OOB_TAG(_tag, _rank, _comm)                             \
  ((((uint64_t)(_tag)) << TORCH_UCX_OOB_BITS_OFFSET) |                         \
   (((uint64_t)(_rank)) << TORCH_UCX_RANK_BITS_OFFSET) |                       \
   (((uint64_t)(_rank)) << TORCH_UCX_COMM_BITS_OFFSET))

#define TORCH_UCX_MAKE_SEND_TAG(_ucp_tag, _tag, _rank, _comm)                  \
  do {                                                                         \
    (_ucp_tag) = TORCH_UCX_MAKE_P2P_TAG((_tag), (_rank), (_comm));             \
  } while (0)

#define TORCH_UCX_ANY_SOURCE (TORCH_UCX_MAX_RANK - 1)
#define TORCH_UCX_ANY_SOURCE_MASK (~TORCH_UCX_RANK_MASK)
#define TORCH_UCX_SPECIFIC_SOURCE_MASK ((uint64_t)-1)

#define TORCH_UCX_MAKE_RECV_TAG(_ucp_tag, _ucp_tag_mask, _tag, _rank, _comm)   \
  do {                                                                         \
    (_ucp_tag) = TORCH_UCX_MAKE_P2P_TAG((_tag), (_rank), (_comm));             \
    if ((_rank) == TORCH_UCX_ANY_SOURCE) {                                     \
      (_ucp_tag_mask) = TORCH_UCX_ANY_SOURCE_MASK;                             \
    } else {                                                                   \
      (_ucp_tag_mask) = TORCH_UCX_SPECIFIC_SOURCE_MASK;                        \
    }                                                                          \
  } while (0)

#define TORCH_UCX_MAKE_OOB_SEND_TAG(_ucp_tag, _tag, _rank, _comm)              \
  do {                                                                         \
    (_ucp_tag) = TORCH_UCX_MAKE_OOB_TAG((_tag), (_rank), (_comm));             \
  } while (0)

#define TORCH_UCX_MAKE_OOB_RECV_TAG(_ucp_tag, _ucp_tag_mask, _tag, _rank,      \
                                    _comm)                                     \
  do {                                                                         \
    (_ucp_tag) = TORCH_UCX_MAKE_OOB_TAG((_tag), (_rank), (_comm));             \
    (_ucp_tag_mask) = (uint64_t)-1;                                            \
  } while (0)

class WorkData {
public:
  // TODO enable this
  // std::vector<at::Tensor> src;
  // std::vector<at::Tensor> dst;
  WorkData() {}
  virtual ~WorkData() = default;
};

class AlltoallWorkData : public WorkData {
public:
  AlltoallWorkData(int size)
      : send_lengths(size), send_offsets(size), recv_lengths(size),
        recv_offsets(size) {}
  std::vector<uint32_t> send_lengths;
  std::vector<uint32_t> send_offsets;
  std::vector<uint32_t> recv_lengths;
  std::vector<uint32_t> recv_offsets;
};

hipStream_t getStreamFromPool(int dev) {
  // TODO
  return 0;
}

hipStream_t getCurrentCUDAStream(int dev) {
  // TODO
  return 0;
}

struct torch_ucc_oob_coll_info_t {
  std::shared_ptr<Store> store;
  uint32_t comm_id;
  int rank;
  int size;
  void *rbuf;
  size_t msglen;
  std::string getKey(std::string key) { return std::to_string(comm_id) + key; }
};

class CommBase {
public:
  CommBase() {}
  virtual void progress() = 0;
  virtual ~CommBase() {}
};

class CommUCX : public CommBase {
public:
  ucp_context_h context;
  ucp_worker_h worker;

public:
  void progress();
  CommUCX(int comm_size);
  ~CommUCX();
};

CommUCX::CommUCX(int comm_size) {
  ucp_params_t params;
  ucp_config_t *config;
  ucs_status_t st;
  ucp_worker_params_t worker_params;

  st = ucp_config_read("TORCH", nullptr, &config);
  check(st == UCS_OK,
        std::string("failed to read UCP config: ") + ucs_status_string(st));
  memset(&params, 0, sizeof(ucp_params_t));
  params.field_mask =
      UCP_PARAM_FIELD_FEATURES | UCP_PARAM_FIELD_REQUEST_SIZE |
      UCP_PARAM_FIELD_ESTIMATED_NUM_EPS | UCP_PARAM_FIELD_TAG_SENDER_MASK |
      UCP_PARAM_FIELD_REQUEST_INIT | UCP_PARAM_FIELD_REQUEST_CLEANUP;
  params.request_size = sizeof(ucc_coll_req_t);
  params.features = UCP_FEATURE_TAG;
  params.estimated_num_eps = comm_size;
  params.tag_sender_mask = TORCH_UCX_RANK_MASK;
  params.request_init = [](void *request) {
    static_cast<ucc_coll_req_h>(request)->status = UCC_INPROGRESS;
  };
  params.request_cleanup = [](void *) {};
  st = ucp_init(&params, config, &context);
  ucp_config_release(config);
  check(st == UCS_OK,
        std::string("failed to init UCP context: ") + ucs_status_string(st));
  memset(&worker_params, 0, sizeof(ucp_worker_params_t));
  worker_params.field_mask = UCP_WORKER_PARAM_FIELD_THREAD_MODE;
  worker_params.thread_mode = UCS_THREAD_MODE_MULTI;
  st = ucp_worker_create(context, &worker_params, &worker);
  check(st == UCS_OK,
        std::string("failed to create UCP worker: ") + ucs_status_string(st));
  ucp_cleanup(context);
}

void CommUCX::progress() { ucp_worker_progress(worker); }

CommUCX::~CommUCX() {
  ucp_worker_destroy(worker);
  ucp_cleanup(context);
}

class CommUCC : public CommBase {
public:
  ucc_lib_h lib;
  ucc_context_h context;

public:
  void progress();
  CommUCC(torch_ucc_oob_coll_info_t *oob_info);
  ~CommUCC();
};

ucc_status_t oob_allgather(void *sbuf, void *rbuf, size_t msglen,
                           void *coll_info, void **req) {
  torch_ucc_oob_coll_info_t *info =
      reinterpret_cast<torch_ucc_oob_coll_info_t *>(coll_info);
  std::vector<char> val =
      std::vector<char>(reinterpret_cast<char *>(sbuf),
                           reinterpret_cast<char *>(sbuf) + msglen);
  info->store->set(info->getKey("teamr" + std::to_string(info->rank)), val);
  info->rbuf = rbuf;
  info->msglen = msglen;
  *req = coll_info;
  return UCC_OK;
}

ucc_status_t oob_allgather_test(void *req) {
  torch_ucc_oob_coll_info_t *info =
      reinterpret_cast<torch_ucc_oob_coll_info_t *>(req);

  for (int r = 0; r < info->size; r++) {
    if (!info->store->check({info->getKey("teamr" + std::to_string(r))})) {
      return UCC_INPROGRESS;
    }
  }
  for (int r = 0; r < info->size; r++) {
    std::vector<char> data =
        info->store->get(info->getKey("teamr" + std::to_string(r)));
    memcpy((void *)((ptrdiff_t)info->rbuf + info->msglen * r), data.data(),
           info->msglen);
  }
  return UCC_OK;
}

ucc_status_t oob_allgather_free(void *req) {
  // torch_ucc_oob_coll_info_t *info =
  //     reinterpret_cast<torch_ucc_oob_coll_info_t *>(req);
  // int num_done = info->store->add({info->getKey("ag_done")}, 1);
  // if (num_done == info->size) {
  //   info->store->deleteKey(info->getKey("ag_done"));
  //   for (int r = 0; r < info->size; r++) {
  //     info->store->deleteKey(info->getKey("teamr" + std::to_string(r)));
  //   }
  //   for (int r = 0; r < info->size; r++) {
  //     info->store->add({info->getKey("ag_free" + std::to_string(r))}, 1);
  //   }
  // } else {
  //   info->store->wait({info->getKey("ag_free" +
  //   std::to_string(info->rank))});
  // }
  // info->store->deleteKey(info->getKey("ag_free" +
  // std::to_string(info->rank)));
  return UCC_OK;
}

CommUCC::CommUCC(torch_ucc_oob_coll_info_t *oob_info) {
  ucc_lib_config_h lib_config;
  ucc_context_config_h context_config;
  ucc_lib_params_t lib_params;
  ucc_context_params_t context_params;
  ucc_status_t st;

  st = ucc_lib_config_read("TORCH", nullptr, &lib_config);
  check(st == UCC_OK,
        std::string("failed to read UCC lib config: ") + ucc_status_string(st));
  memset(&lib_params, 0, sizeof(ucc_lib_params_t));
  lib_params.mask = UCC_LIB_PARAM_FIELD_THREAD_MODE;
  lib_params.thread_mode = UCC_THREAD_MULTIPLE;
  st = ucc_init(&lib_params, lib_config, &lib);
  ucc_lib_config_release(lib_config);
  check(st == UCC_OK,
        std::string("failed to init UCC lib: ") + ucc_status_string(st));
  ucc_lib_attr_t lib_attr;
  lib_attr.mask = UCC_LIB_ATTR_FIELD_THREAD_MODE;
  st = ucc_lib_get_attr(lib, &lib_attr);
  check(st == UCC_OK,
        std::string("failed to query for lib attr: ") + ucc_status_string(st));
  check(lib_attr.thread_mode == UCC_THREAD_MULTIPLE,
        "ucc library wasn't initialized with mt support "
        "check ucc compile options ");
  st = ucc_context_config_read(lib, NULL, &context_config);
  check(st == UCC_OK, std::string("failed to read UCC context config: ") +
                          ucc_status_string(st));
  st = ucc_context_config_modify(context_config, NULL, "ESTIMATED_NUM_EPS",
                                 std::to_string(oob_info->size).c_str());
  check(st == UCC_OK, std::string("failed to modify UCC context config: ") +
                          ucc_status_string(st));
  memset(&context_params, 0, sizeof(ucc_context_params_t));
  context_params.mask =
      UCC_CONTEXT_PARAM_FIELD_TYPE | UCC_CONTEXT_PARAM_FIELD_OOB;
  context_params.type = UCC_CONTEXT_SHARED;
  context_params.oob.participants = oob_info->size;
  context_params.oob.allgather = oob_allgather;
  context_params.oob.req_test = oob_allgather_test;
  context_params.oob.req_free = oob_allgather_free;
  context_params.oob.coll_info = oob_info;
  ucc_context_create(lib, &context_params, context_config, &context);
  ucc_context_config_release(context_config);
  check(st == UCC_OK,
        std::string("failed to create UCC context: ") + ucc_status_string(st));
}

void CommUCC::progress() { ucc_context_progress(context); }

CommUCC::~CommUCC() {
  ucc_context_destroy(context);
  ucc_finalize(lib);
}

enum torch_ucx_tag_type_t { TORCH_UCX_P2P_TAG, TORCH_UCX_OOB_TAG };

struct event_pool_t {
  std::queue<std::unique_ptr<hipEvent_t>> event_pool;
  std::mutex event_pool_mutex;
};

class WorkUCC {
  OpType opType;
  friend class CommPG;

public:
  WorkUCC(OpType opType, ucc_status_t status, ucc_coll_req_h request,
          ucc_ee_h ee, CommBase *comm)
      : opType(opType), status_(status), request_(request), comm_(comm) {}
  ~WorkUCC();
  bool isCompleted();
  bool isSuccess() const;
  bool wait(std::chrono::milliseconds timeout = kUnsetTimeout);
  void finalize();
  std::unique_ptr<WorkData> data;
  std::unique_ptr<hipEvent_t> fence = nullptr;
  event_pool_t *ep = nullptr;

protected:
  ucc_status_t status_;
  ucc_coll_req_h request_;
  CommBase *comm_;
};

WorkUCC::~WorkUCC() {
  check(request_ == nullptr, "TorchUCC, request wasn't finalized");
  if (fence && ep) {
    std::lock_guard<std::mutex> lock(ep->event_pool_mutex);
    ep->event_pool.push(std::move(fence));
  }
}

void WorkUCC::finalize() {
  if (request_ != nullptr) {
    if (isP2POp(opType)) {
      request_->status = UCC_INPROGRESS;
      ucp_request_free(request_);
    } else {
      ucc_collective_finalize(request_);
    }
    status_ = UCC_OK;
    request_ = nullptr;
  }
}

class CommPG {
  CommUCX ucx_comm;
  CommUCC ucc_comm;
  int device_index;
  std::mutex mutex;
  std::thread progress_thread;
  std::condition_variable queue_produce_cv;
  std::condition_variable queue_consume_cv;
  std::deque<std::shared_ptr<WorkUCC>> progress_queue;
  bool stop_progress_loop;

public:
  int cuda_device_index;
  CommPG(torch_ucc_oob_coll_info_t *oob_info, int dev);

  ~CommPG();

  void ucx_connect_eps(std::vector<ucp_ep_h> &eps,
                       torch_ucc_oob_coll_info_t *oob);

  void ucx_disconnect_eps(std::vector<ucp_ep_h> &eps,
                          torch_ucc_oob_coll_info_t *oob);

  void ucc_create_team(ucc_team_h &team, torch_ucc_oob_coll_info_t *oob_info);

  void ucc_destroy_team(ucc_team_h &team);

  std::shared_ptr<WorkUCC> enqueue_p2p(OpType opType, ucc_coll_req_h request);

  std::shared_ptr<WorkUCC>
  enqueue_cuda_collective(OpType opType, ucc_coll_args_t &coll,
                          std::unique_ptr<WorkData> data, ucc_team_h &team,
                          ucc_ee_h ee, std::unique_ptr<hipEvent_t> cuda_ev,
                          const hipStream_t &stream, event_pool_t *ep);

  std::shared_ptr<WorkUCC> enqueue_collective(OpType opType,
                                              ucc_coll_args_t &coll,
                                              std::unique_ptr<WorkData> data,
                                              ucc_team_h &team);

  static std::shared_ptr<CommPG> get_comm(uint32_t &id, int dev,
                                          torch_ucc_oob_coll_info_t *oob);

  void progress_loop();

  ucc_coll_req_h send_nb(ucp_ep_h ep, void *data, ucs_memory_type_t mtype,
                         size_t size, ucp_tag_t ucp_tag);

  ucc_coll_req_h recv_nb(void *data, ucs_memory_type_t mtype, size_t size,
                         ucp_tag_t ucp_tag, ucp_tag_t ucp_tag_mask);
};

CommPG::CommPG(torch_ucc_oob_coll_info_t *oob_info, int dev)
    : ucx_comm(oob_info->size), ucc_comm(oob_info), cuda_device_index(dev) {
  stop_progress_loop = false;
  progress_thread = std::thread(&CommPG::progress_loop, this);
  pthread_setname_np(progress_thread.native_handle(), "ucc-progress");
}

CommPG::~CommPG() {
  std::unique_lock<std::mutex> lock(mutex);
  queue_consume_cv.wait(lock, [&] { return progress_queue.empty(); });
  stop_progress_loop = true;
  lock.unlock();
  queue_produce_cv.notify_all();
  progress_thread.join();
}

std::shared_ptr<CommPG> CommPG::get_comm(uint32_t &id, int dev,
                                         torch_ucc_oob_coll_info_t *oob) {
  static std::mutex m;
  static std::weak_ptr<CommPG> comm;
  static uint32_t comm_id;

  std::lock_guard<std::mutex> lock(m);
  id = (comm_id++ % TORCH_UCX_COMM_BITS);
  oob->comm_id = id;
  std::shared_ptr<CommPG> shared_comm = comm.lock();
  if (!shared_comm) {
    shared_comm = std::make_shared<CommPG>(oob, dev);
    comm = shared_comm;
  } else {
    check((shared_comm->cuda_device_index == TORCH_UCC_DEVICE_NOT_SET) ||
              (shared_comm->cuda_device_index == dev),
          "ucc communicator was initialized with different cuda device,"
          "multi device is not supported");
    shared_comm->cuda_device_index = dev;
  }
  return shared_comm;
}

void CommPG::ucx_connect_eps(std::vector<ucp_ep_h> &eps,
                             torch_ucc_oob_coll_info_t *oob) {
  ucs_status_t st;
  ucp_address_t *local_addr;
  size_t local_addr_len;
  std::vector<char> peer_addr;

  st = ucp_worker_get_address(ucx_comm.worker, &local_addr, &local_addr_len);
  check(st == UCS_OK, "failed to get worker address");
  std::vector<char> val =
      std::vector<char>(reinterpret_cast<char *>(local_addr),
                        reinterpret_cast<char *>(local_addr) + local_addr_len);
  oob->store->set(oob->getKey("wa" + std::to_string(oob->rank)), val);
  ucp_worker_release_address(ucx_comm.worker, local_addr);
  eps.resize(oob->size);
  for (int i = 0; i < oob->size; i++) {
    peer_addr = oob->store->get(oob->getKey("wa" + std::to_string(i)));
    ucp_ep_params_t ep_params;
    ep_params.field_mask = UCP_EP_PARAM_FIELD_REMOTE_ADDRESS;
    ep_params.address = reinterpret_cast<ucp_address_t *>(peer_addr.data());
    st = ucp_ep_create(ucx_comm.worker, &ep_params, &(eps[i]));
    check(st == UCS_OK, "failed to create endpoint");
  }
}

// TODO: remove this?
// void CommPG::ucx_disconnect_eps(std::vector<ucp_ep_h> &eps,
//                                 torch_ucc_oob_coll_info_t *oob) {
//   ucs_status_t st;

//   for (ucp_ep_h &ep : eps) {
//     ucs_status_ptr_t close_req = ucp_ep_close_nb(ep,
//     UCP_EP_CLOSE_MODE_FLUSH); check(!UCS_PTR_IS_ERR(close_req), "failed to
//     close endpoint"); if (UCS_PTR_IS_PTR(close_req)) {
//       do {
//         ucp_worker_progress(ucx_comm.worker);
//         st = ucp_request_check_status(close_req);
//       } while (st != UCS_OK);
//       ucp_request_free(close_req);
//     }
//   }
//   if ((size_t)oob->store->add(oob->getKey("epclosed"), 1) == eps.size()) {
//     oob->store->add(oob->getKey("epfinished"), 1);
//   } else {
//     oob->store->wait({oob->getKey("epfinished")});
//   }
// }

// TODO: can I delete this?
// ucc_coll_req_h CommPG::send_nb(ucp_ep_h ep, void *data, ucs_memory_type_t
// mtype,
//                                size_t size, ucp_tag_t ucp_tag) {
//   ucs_status_ptr_t st;
//   ucp_request_param_t params;
//   params.op_attr_mask = UCP_OP_ATTR_FIELD_CALLBACK |
//                         UCP_OP_ATTR_FIELD_DATATYPE |
//                         UCP_OP_ATTR_FIELD_MEMORY_TYPE;
//   params.datatype = ucp_dt_make_contig(size);
//   params.memory_type = mtype;
//   params.cb.send = [](void *request, ucs_status_t status, void *user_data) {
//     static_cast<ucc_coll_req_h>(request)->status = UCC_OK;
//   };
//   st = ucp_tag_send_nbx(ep, data, 1, ucp_tag, &params);
//   check(!UCS_PTR_IS_ERR(st), std::string("failed to send message: ") +
//                                  ucs_status_string(UCS_PTR_STATUS(st)));
//   return reinterpret_cast<ucc_coll_req_h>(st);
// }

// TODO: can I delete this?
// ucc_coll_req_h CommPG::recv_nb(void *data, ucs_memory_type_t mtype, size_t
// size,
//                                ucp_tag_t ucp_tag, ucp_tag_t ucp_tag_mask) {
//   ucs_status_ptr_t st;
//   ucp_request_param_t params;
//   params.op_attr_mask = UCP_OP_ATTR_FIELD_CALLBACK |
//                         UCP_OP_ATTR_FIELD_DATATYPE |
//                         UCP_OP_ATTR_FIELD_MEMORY_TYPE;
//   params.datatype = ucp_dt_make_contig(size);
//   params.cb.recv = [](void *request, ucs_status_t status,
//                       const ucp_tag_recv_info_t *info, void *user_data) {
//     static_cast<ucc_coll_req_h>(request)->status = UCC_OK;
//   };
//   params.memory_type = mtype;
//   st = ucp_tag_recv_nbx(ucx_comm.worker, data, 1, ucp_tag, ucp_tag_mask,
//                         &params);
//   check(!UCS_PTR_IS_ERR(st), std::string("failed to recv message: ") +
//                                  ucs_status_string(UCS_PTR_STATUS(st)));
//   return reinterpret_cast<ucc_coll_req_h>(st);
// }

void CommPG::ucc_create_team(ucc_team_h &team,
                             torch_ucc_oob_coll_info_t *oob_info) {
  ucc_status_t st;
  ucc_team_params_t team_params;
  team_params.mask = UCC_TEAM_PARAM_FIELD_EP | UCC_TEAM_PARAM_FIELD_EP_RANGE |
                     UCC_TEAM_PARAM_FIELD_OOB;
  team_params.oob.allgather = oob_allgather;
  team_params.oob.req_test = oob_allgather_test;
  team_params.oob.req_free = oob_allgather_free;
  team_params.oob.coll_info = oob_info;
  team_params.oob.participants = oob_info->size;
  team_params.ep = oob_info->rank;
  team_params.ep_range = UCC_COLLECTIVE_EP_RANGE_CONTIG;
  st = ucc_team_create_post(&ucc_comm.context, 1, &team_params, &team);
  std::cout << "ucc_team_create_post" << std::endl;
  check(st == UCC_OK,
        std::string("failed to post team create: ") + ucc_status_string(st));
  do {
    st = ucc_team_create_test(team);
  } while (st == UCC_INPROGRESS);
  check(st == UCC_OK,
        std::string("failed to create UCC team: ") + ucc_status_string(st));
  std::cout << "ucc_create_team" << std::endl;
}

void CommPG::ucc_destroy_team(ucc_team_h &team) {
  ucc_status_t status;
  while (UCC_INPROGRESS == (status = ucc_team_destroy(team))) {
    check(status == UCC_OK,
          std::string("ucc team destroy error: ") + ucc_status_string(status));
  }
}

std::shared_ptr<WorkUCC> CommPG::enqueue_p2p(OpType opType,
                                             ucc_coll_req_h request) {
  if (request == nullptr) {
    // p2p2 request completed immediately don't save it to progress queue
    return std::make_shared<WorkUCC>(opType, UCC_OK, request, nullptr,
                                     &ucx_comm);
  }
  auto work = std::make_shared<WorkUCC>(opType, UCC_INPROGRESS, request,
                                        nullptr, &ucx_comm);
  std::unique_lock<std::mutex> lock(mutex);
  progress_queue.push_back(work);
  lock.unlock();
  queue_produce_cv.notify_one();
  return work;
}

std::shared_ptr<WorkUCC>
CommPG::enqueue_collective(OpType opType, ucc_coll_args_t &coll,
                           std::unique_ptr<WorkData> data, ucc_team_h &team) {
  ucc_coll_req_h request;
  ucc_status_t st;
  st = ucc_collective_init(&coll, &request, team);
  check(st == UCC_OK,
        std::string("failed to init collective: ") + ucc_status_string(st));
  st = ucc_collective_post(request);
  check(st == UCC_OK,
        std::string("failed to post collective: ") + ucc_status_string(st));
  auto work = std::make_shared<WorkUCC>(opType, UCC_INPROGRESS, request,
                                        nullptr, &ucc_comm);
  work->data = std::move(data);
  std::unique_lock<std::mutex> lock(mutex);
  progress_queue.push_back(work);
  lock.unlock();
  queue_produce_cv.notify_one();
  return work;
}

std::shared_ptr<WorkUCC> CommPG::enqueue_cuda_collective(
    OpType opType, ucc_coll_args_t &coll, std::unique_ptr<WorkData> data,
    ucc_team_h &team, ucc_ee_h ee, std::unique_ptr<hipEvent_t> cuda_ev,
    const hipStream_t &stream, event_pool_t *ep) {
  ucc_coll_req_h request;
  ucc_status_t st;
  st = ucc_collective_init(&coll, &request, team);
  check(st == UCC_OK,
        std::string("failed to init collective: ") + ucc_status_string(st));
  ucc_ev_t comp_ev, *post_ev;
  comp_ev.ev_type = UCC_EVENT_COMPUTE_COMPLETE;
  comp_ev.ev_context = nullptr;
  comp_ev.ev_context_size = 0;
  comp_ev.req = request;
  st = ucc_collective_triggered_post(ee, &comp_ev);
  check(st == UCC_OK, std::string("failed to post triggered collective: ") +
                          ucc_status_string(st));
  st = ucc_ee_get_event(ee, &post_ev);
  check(st == UCC_OK && post_ev->ev_type == UCC_EVENT_COLLECTIVE_POST,
        "Bug???");
  ucc_ee_ack_event(ee, post_ev);
  auto work =
      std::make_shared<WorkUCC>(opType, UCC_INPROGRESS, request, ee, &ucc_comm);
  work->data = std::move(data);
  work->ep = ep;
  check_cuda(hipEventRecord(*cuda_ev, stream));
  work->fence = std::move(cuda_ev);
  std::unique_lock<std::mutex> lock(mutex);
  progress_queue.push_back(work);
  lock.unlock();
  queue_produce_cv.notify_one();
  return work;
}

void CommPG::progress_loop() {
  std::unique_lock<std::mutex> lock(mutex);
  bool device_set = false;
  while (!stop_progress_loop) {
    if (progress_queue.empty()) {
      queue_produce_cv.wait(lock);
      continue;
    }
    auto work = progress_queue.front();
    progress_queue.pop_front();
    lock.unlock();
    queue_consume_cv.notify_one();
    if ((!device_set) && (cuda_device_index != TORCH_UCC_DEVICE_NOT_SET)) {
      set_device(cuda_device_index);
      device_set = true;
    }
    while (work->request_->status > 0) {
      // operation initialized is in progress or
      work->comm_->progress();
    }
    work->finalize();
    work->data.reset();
    lock.lock();
  }
}

torch_ucc_oob_coll_info_t oob;
std::shared_ptr<CommPG> comm;
uint32_t comm_id;
std::vector<ucp_ep_h> eps;
ucc_team_h team;
ucc_ee_h cuda_ee;
std::shared_ptr<hipStream_t> stream =
    nullptr; // TODO, it was unique_ptr in its original code
event_pool_t ep;

void initProcessGroupUCC(const std::shared_ptr<Store> &store, int rank,
                         int size) {
  // TODO: should size be world size?
  oob.rank = rank;
  oob.size = size;
  oob.store = store;
  comm = nullptr;
  cuda_ee = nullptr;
}

void initComm(int dev) {
  if (!comm) {
    set_device(dev);
    comm = CommPG::get_comm(comm_id, dev, &oob);
    comm->ucx_connect_eps(eps, &oob);
    comm->ucc_create_team(team, &oob);
  } else {
    check((comm->cuda_device_index == TORCH_UCC_DEVICE_NOT_SET) ||
              (comm->cuda_device_index == dev),
          "ucc communicator was initialized with different cuda device, "
          "multi device is not supported");
    comm->cuda_device_index = dev;
  }
  if (!cuda_ee) {
    ucc_status_t st;
    stream = std::make_shared<hipStream_t>(getStreamFromPool(dev));
    ucc_ee_params_t params;
    params.ee_type = UCC_EE_CUDA_STREAM;
    params.ee_context = (void *)stream.get();
    params.ee_context_size = sizeof(hipStream_t);
    st = ucc_ee_create(team, &params, &cuda_ee);
    check(st == UCC_OK,
          std::string("failed to create UCC EE: ") + ucc_status_string(st));
  }
}

std::shared_ptr<WorkUCC> collective_post(OpType opType, ucc_coll_args_t &coll,
                                         std::unique_ptr<WorkData> data,
                                         int dev) {
  std::unique_ptr<hipEvent_t> cuda_ev;
  {
    std::lock_guard<std::mutex> lock(ep.event_pool_mutex);
    if (ep.event_pool.empty()) {
      cuda_ev = std::make_unique<hipEvent_t>();
    } else {
      cuda_ev = std::move(ep.event_pool.front());
      ep.event_pool.pop();
    }
  }
  auto current_stream = getCurrentCUDAStream(dev);
  check_cuda(hipEventRecord(*cuda_ev, current_stream));
  check_cuda(hipStreamWaitEvent(*stream, *cuda_ev));
  auto work =
      comm->enqueue_cuda_collective(opType, coll, std::move(data), team,
                                    cuda_ee, std::move(cuda_ev), *stream, &ep);
  return work;
}

std::shared_ptr<WorkUCC> alltoall() {
  initProcessGroupUCC({}, rank, world_size); // TODO use a real store
  initComm(get_device());

  // TODO initialize them
  std::vector<int64_t> outputSplitSizes;
  std::vector<int64_t> inputSplitSizes;
  ucc_coll_args_t coll;
  AlltoallWorkData *data;

  if ((outputSplitSizes.size() == 0) && (inputSplitSizes.size() == 0)) {
    data = new AlltoallWorkData(0);
    // TODO: migrate this
    // TORCH_CHECK((outputTensor.size(0) % size_ == 0) &&
    //                 (inputTensor.size(0) % size_ == 0),
    //             "Tensor's dim 0 does not divide equally across group size");
    coll.mask = 0;
    coll.coll_type = UCC_COLL_TYPE_ALLTOALL;
    // TODO: enable this
    // coll.src.info.buffer = inputTensor.data_ptr();
    // coll.src.info.count =
    //     inputTensor.element_size() * inputTensor.numel() / size_;
    coll.src.info.datatype = UCC_DT_UINT8;
    coll.src.info.mem_type = UCC_MEMORY_TYPE_CUDA;
    // TODO: enable this
    // coll.dst.info.buffer = outputTensor.data_ptr();
    // coll.dst.info.count =
    //     outputTensor.element_size() * outputTensor.numel() / size_;
    coll.dst.info.datatype = UCC_DT_UINT8;
    coll.dst.info.mem_type = UCC_MEMORY_TYPE_CUDA;
  } else {
    data = new AlltoallWorkData(size_);
    // TODO: migrate this
    // c10d::checkSplitSizes(inputSplitSizes, inputTensor, size_);
    // c10d::checkSplitSizes(outputSplitSizes, outputTensor, size_);
    // computeLengthsAndOffsets(outputSplitSizes, outputTensor,
    //                          &data->recv_lengths, &data->recv_offsets);
    // computeLengthsAndOffsets(inputSplitSizes, inputTensor,
    // &data->send_lengths,
    //                          &data->send_offsets);
    coll.mask = UCC_COLL_ARGS_FIELD_FLAGS;
    coll.coll_type = UCC_COLL_TYPE_ALLTOALLV;
    // TODO: enable this
    // coll.src.info_v.buffer = inputTensor.data_ptr();
    coll.src.info_v.counts = (ucc_count_t *)data->send_lengths.data();
    coll.src.info_v.displacements = (ucc_aint_t *)data->send_offsets.data();
    coll.src.info_v.datatype = dtype;
    coll.src.info_v.mem_type = UCC_MEMORY_TYPE_CUDA;
    // TODO: enable this
    // coll.dst.info_v.buffer = outputTensor.data_ptr();
    coll.dst.info_v.counts = (ucc_count_t *)data->recv_lengths.data();
    coll.dst.info_v.displacements = (ucc_aint_t *)data->recv_offsets.data();
    coll.dst.info_v.datatype = dtype;
    coll.dst.info_v.mem_type = UCC_MEMORY_TYPE_CUDA;
    coll.flags = UCC_COLL_ARGS_FLAG_CONTIG_SRC_BUFFER |
                 UCC_COLL_ARGS_FLAG_CONTIG_DST_BUFFER;
  }
  // TODO: enable this
  // data->src = {inputTensor};
  // data->dst = {outputTensor};
  return collective_post(OpType::ALLTOALL_BASE, coll,
                         std::unique_ptr<WorkData>(data), get_device());
}
#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <random>
#include <stdexcept>

#include "utils.hpp"

int N = 5;
using T = int;
int world_size;
int rank;

void check_cuda(hipError_t err) {
  check(err == hipSuccess, hipGetErrorString(err));
}

void check_cuda() {
  check_cuda(hipGetLastError());
}

int get_device() {
  return rank;
}

void set_device() {
  check_cuda(hipSetDevice(get_device()));
}

void set_device(int i) {
  check_cuda(hipSetDevice(i));
}

T *input;
T *output;

void allocate_buffers() {
  check_cuda(hipMalloc(&input, sizeof(T) * N * world_size));
  check_cuda(hipMalloc(&output, sizeof(T) * N * world_size));
}

__global__ void write_value(T *ptr, T value) {
  *ptr = value;
}

void initialize_input() {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_int_distribution<int> d(0, N * world_size);

  for (int i = 0; i < world_size; i++) {
    for (int j = 0; j < N; j++) {
      T value = (i == rank ? d(gen): 0);
      write_value<<<1, 1>>>(input + N * i + j, value);
    }
  }
}

template<typename T>
void print_buffer(T *ptr) {
  hipDeviceSynchronize();
  for (int i = 0; i < world_size; i++) {
    T *host = new T[N];
    check_cuda(hipMemcpy(host, ptr + N * i, sizeof(T) * N, hipMemcpyDefault));
    for (int j = 0; j < N; j++) {
      std::cout << host[j] << ", ";
    }
    delete [] host;
    std::cout << std::endl;
  }
}

void alltoall();

int main(int argc, char *argv[]) {
  check(argc == 3, "Bad argument");
  world_size = std::stoi(argv[1]);
  rank = std::stoi(argv[2]);
  std::cout << "World size: " << world_size << ", " << "Rank: " << rank << std::endl;

  set_device();

  allocate_buffers();
  initialize_input();
  std::cout << std::endl << "Buffers initialized as:" << std::endl;
  print_buffer(input);

  alltoall();

  std::cout << std::endl << "After alltoall, buffers are:" << std::endl;
  print_buffer(output);
}